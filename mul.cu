
#include <hip/hip_runtime.h>
#include<iostream>
#include<time.h>
#include<cstdlib>
#include<stdlib.h>

using namespace std;
__global__ void matrixMultiplication(int* A,int* B,int* C,int N);


void mm(int* A,int* B,int* C,int N);

int main()
{
	int ROWS = 1<<2;
	int COLS = 1<<2; //since in question it is asked to take N*N array

	int* hostA = (int*)malloc(sizeof(int)*ROWS*COLS);
	int* hostB = (int*)malloc(sizeof(int)*ROWS*COLS);
	int* hostC = (int*)malloc(sizeof(int)*ROWS*COLS);


	//initialize matrices A and B by random numbers
	srand(time(0));
	int i,j;
	for(i=0;i<ROWS;i++)
	{
		for(j=0;j<COLS;j++)
		{
			hostB[i*COLS+j] = rand()%30;
			hostA[i*COLS+j] = rand()%20;
		}
	}

	int* deviceA,*deviceB,*deviceC;

	hipMalloc(&deviceA,sizeof(int)*ROWS*COLS);
	hipMalloc(&deviceB,sizeof(int)*ROWS*COLS);
	hipMalloc(&deviceC,sizeof(int)*ROWS*COLS);

	hipMemcpy(deviceA,hostA,sizeof(int)*ROWS*COLS,hipMemcpyHostToDevice);
	
	hipMemcpy(deviceB,hostB,sizeof(int)*ROWS*COLS,hipMemcpyHostToDevice);

	mm(deviceA,deviceB,deviceC,ROWS);


	hipError_t e=hipGetLastError();                                 
 if(e!=hipSuccess) {                                              
   printf("Cuda failure %s: ",hipGetErrorString(e));           
   
 }     

	hipDeviceSynchronize();
   
    hipMemcpy(hostC,deviceC,ROWS*COLS*sizeof(int),hipMemcpyDeviceToHost);


	hipFree(deviceA);
	hipFree(deviceB);
	hipFree(deviceC);


	//now do actual multiplication
	int N = ROWS;
	int* actual = (int*)malloc(sizeof(int)*ROWS*COLS);
	int sum;
	for (int row=0; row<ROWS; row++){
        for (int col=0; col<COLS; col++){
             sum=0;
            for (int n=0; n<N; n++){
                sum += hostA[row*N+n]*hostB[n*N+col];
            }
            actual[row*N+col] = sum;
        }
    }


    double error = 0;
    for(int k=0;k<ROWS*COLS;k++)
	{
	cout<<k<<")"<< "Expected value = "<<actual[k]<<" Actual value = "<<hostC[k]<<"\n";
	error += double(abs(actual[k]-hostC[k]));
	}




	error=sqrt(error);
	cout<<"error = "<<error<<"\n";

	delete[] hostA;
    delete[] hostB;
    delete[] hostC;



}

__global__ void matrixMultiplication(int* A,int* B,int* C,int N)
{

	//printf("hi in mult\n");
	int ROW = blockIdx.y*blockDim.y+threadIdx.y;
	int COL = blockIdx.x*blockDim.x+threadIdx.x;

	   

    //printf("ROW = %d and COL is %d\n",ROW,COL);
	int sum =0 ;
	if(ROW<N && COL<N)
	{
	for(int i=0;i<N;i++)
	{
	sum+=A[ROW*N+i]*B[i*N+COL];
	}
	__syncthreads();    
	C[ROW*N+COL]=sum;
	}
	
}


void mm(int* A,int* B,int* C,int N)
{
	dim3 threadsPerblock(N,N);
	dim3 blocksPerGrid(1,1);

	if(N*N>512)
	{
	threadsPerblock.x = 512;
	threadsPerblock.y=512;
	blocksPerGrid.x = ceil(double(N)/double(threadsPerblock.x));
	blocksPerGrid.y = ceil(double(N)/double(threadsPerblock.y));

	}

	cout<<"calling mult"<<"\n";

	matrixMultiplication<<<blocksPerGrid,threadsPerblock>>>(A,B,C,N);
}